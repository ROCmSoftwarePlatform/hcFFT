#include <stdio.h>
#include <assert.h>
// CUDA runtime
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#define NX 16
#define BATCH 1

int main()
{
	hipfftHandle plan;
	hipfftComplex *input = (hipfftComplex*)calloc(NX, sizeof(hipfftComplex));
	hipfftComplex *output = (hipfftComplex*)calloc(NX, sizeof(hipfftComplex));
	hipfftComplex *idata;
	hipfftComplex *odata;

	int seed = 123456789;
	srand(seed);

	for(int i=0; i<NX; i++)
	{
		input[i].x = rand();
		input[i].y = rand();
	}

	hipMalloc((void**)&idata, sizeof(hipfftComplex)*NX*BATCH);
	hipMemcpy(idata, input, sizeof(hipfftComplex)*NX*BATCH, hipMemcpyHostToDevice);
	hipMalloc((void**)&odata, sizeof(hipfftComplex)*NX*BATCH);
	hipMemcpy(odata, output, sizeof(hipfftComplex)*NX*BATCH, hipMemcpyHostToDevice);
	

	if (hipGetLastError() != hipSuccess){
		fprintf(stderr, "Cuda error: Failed to allocate\n");
		return 0;	
	}

	if (hipfftPlan1d(&plan, NX, HIPFFT_C2C, BATCH) != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT error: Plan creation failed");
		return 0;	
	}	

	if (hipfftExecC2C(plan, (hipfftComplex*)idata, (hipfftComplex *)odata, HIPFFT_FORWARD) != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT error: ExecC2C Forward failed");
		return 0;	
	}

	if (hipfftExecC2C(plan,(hipfftComplex*) idata, (hipfftComplex *)odata, HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT error: ExecC2C Inverse failed");
		return 0;	
	}

	if (hipDeviceSynchronize() != hipSuccess){
		fprintf(stderr, "Cuda error: Failed to synchronize\n");
		return 0;	
	}	

	hipfftDestroy(plan);

	free(input);
	free(output);

	hipFree(idata);
	hipFree(odata);

	return 0;
}

