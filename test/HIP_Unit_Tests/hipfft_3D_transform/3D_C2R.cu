#include <stdio.h>
#include <assert.h>
// CUDA runtime
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#define NX 64 
#define NY 128 
#define NZ 128 
#define BATCH 1

int main()
{
	hipfftHandle plan;
	int Csize = ((NX/2) + 1) * NY * NZ;
	int Rsize = NX * NY *NZ;
	hipfftComplex *input = (hipfftComplex*)calloc(Csize, sizeof(hipfftComplex));
	hipfftReal *output = (hipfftReal*)calloc(Rsize, sizeof(hipfftReal));
	hipfftComplex *idata;
	hipfftReal *odata;

	int seed = 123456789;
	srand(seed);

	for(int i=0; i<Csize; i++)
	{
		input[i].x = rand();
		input[i].y = rand();
	}

	hipMalloc((void**)&idata, sizeof(hipfftComplex)*Csize*BATCH);
	hipMemcpy(idata, input, sizeof(hipfftComplex)*Csize*BATCH, hipMemcpyHostToDevice);
	hipMalloc((void**)&odata, sizeof(hipfftReal)*Rsize*BATCH);
	hipMemcpy(odata, output, sizeof(hipfftReal)*Rsize*BATCH, hipMemcpyHostToDevice);

	if (hipGetLastError() != hipSuccess)
	{ 
		fprintf(stderr, "Cuda error: Failed to allocate\n"); 
		return 0;	
	} 

	/* Create a 3D FFT plan. */ 
	if (hipfftPlan3d(&plan, NX, NY, NZ, HIPFFT_C2R) != HIPFFT_SUCCESS) 
	{ 
		fprintf(stderr, "CUFFT error: Plan creation failed"); 
		return 0;	
	}	

	/* Use the CUFFT plan to transform the signal in place. */ 
	if (hipfftExecC2R(plan, (hipfftComplex*)idata, (hipfftReal*)odata) != HIPFFT_SUCCESS)
	{ 
		fprintf(stderr, "CUFFT error: ExecC2R failed"); 
		return 0;	
	}

	if (hipDeviceSynchronize() != hipSuccess)
	{ 
		fprintf(stderr, "Cuda error: Failed to synchronize\n"); 
		return 0;	
	}	
	
	hipfftDestroy(plan); 

	free(input);
	free(output);

	hipFree(idata);
	hipFree(odata); 

	return 0;
}

