#include <stdio.h>
#include <assert.h>
// CUDA runtime
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#define NX 256 
#define NY 128 
#define NRANK 2 
#define BATCH 1

int main()
{
	hipfftHandle plan;
	int Csize = ((NX/2) + 1) * NY;
	int Rsize = NX * NY;
	hipfftDoubleComplex *input = (hipfftDoubleComplex*)calloc(Csize, sizeof(hipfftDoubleComplex));
	hipfftDoubleReal *output = (hipfftDoubleReal*)calloc(Rsize, sizeof(hipfftDoubleReal));
	hipfftDoubleComplex *idata;
	hipfftDoubleReal *odata;

	int seed = 123456789;
	srand(seed);

	for(int i=0; i<Csize; i++)
	{
		input[i].x = rand();
		input[i].y = rand();
	}

	hipMalloc((void**)&idata, sizeof(hipfftDoubleComplex)*Csize*BATCH);
	hipMemcpy(idata, input, sizeof(hipfftDoubleComplex)*Csize*BATCH, hipMemcpyHostToDevice);
	hipMalloc((void**)&odata, sizeof(hipfftDoubleReal)*Rsize*BATCH);
	hipMemcpy(odata, output, sizeof(hipfftDoubleReal)*Rsize*BATCH, hipMemcpyHostToDevice);
	
	if (hipGetLastError() != hipSuccess)
	{ 
		fprintf(stderr, "Cuda error: Failed to allocate\n"); 
		return 0;
	} 

	/* Create a 2D FFT plan. */ 
	if (hipfftPlan2d(&plan, NX, NY, HIPFFT_Z2D) != HIPFFT_SUCCESS)
	{ 
		fprintf(stderr, "CUFFT Error: Unable to create plan\n"); 
		return 0;
	}


	if (hipfftExecZ2D(plan, (hipfftDoubleComplex*)idata, (hipfftDoubleReal*)odata) != HIPFFT_SUCCESS)
	{ 
		fprintf(stderr, "CUFFT error: ExecZ2D failed\n"); 
		return 0;	
	} 
	if (hipDeviceSynchronize() != hipSuccess)
	{ 
		fprintf(stderr, "Cuda error: Failed to synchronize\n"); 
		return 0; 
	}

	hipfftDestroy(plan); 

	free(input);
	free(output);

	hipFree(idata);
	hipFree(odata);

	return 0;
}

