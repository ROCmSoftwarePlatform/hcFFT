#include <stdio.h>
#include <assert.h>
// CUDA runtime
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#define NX 128 
#define NY 128 
#define NZ 128 
#define BATCH 1

int main()
{
	hipfftHandle plan;
	int Csize = ((NX/2) + 1) * NY * NZ;
	int Rsize = NX * NY * NZ;
	hipfftDoubleReal *input = (hipfftDoubleReal*)calloc(Rsize, sizeof(hipfftDoubleReal));
	hipfftDoubleComplex *output = (hipfftDoubleComplex*)calloc(Csize, sizeof(hipfftDoubleComplex));
	hipfftDoubleReal *idata;
	hipfftDoubleComplex *odata;
	
	int seed = 123456789;
	srand(seed);

	for(int i=0; i<Rsize; i++)
	{
		input[i] = rand();
	}

	hipMalloc((void**)&idata, sizeof(hipfftDoubleReal)*Rsize*BATCH);
	hipMemcpy(idata, input, sizeof(hipfftDoubleReal)*Rsize*BATCH, hipMemcpyHostToDevice);
	hipMalloc((void**)&odata, sizeof(hipfftDoubleComplex)*Csize*BATCH);
	hipMemcpy(odata, output, sizeof(hipfftDoubleComplex)*Csize*BATCH, hipMemcpyHostToDevice);	
 
	if (hipGetLastError() != hipSuccess)
	{ 
		fprintf(stderr, "Cuda error: Failed to allocate\n"); 
		return 0;	
	} 

	/* Create a 3D FFT plan. */ 
	if (hipfftPlan3d(&plan, NX, NY, NZ, HIPFFT_D2Z) != HIPFFT_SUCCESS) 
	{ 
		fprintf(stderr, "CUFFT error: Plan creation failed"); 
		return 0;	
	}	

	/* Use the CUFFT plan to transform the signal in place. */ 
	if (hipfftExecD2Z(plan, (hipfftDoubleReal*)idata, (hipfftDoubleComplex*)odata) != HIPFFT_SUCCESS)
	{ 
		fprintf(stderr, "CUFFT error: ExecD2Z failed"); 
		return 0;	
	}

	if (hipDeviceSynchronize() != hipSuccess)
	{ 
		fprintf(stderr, "Cuda error: Failed to synchronize\n"); 
		return 0;	
	}	
	
	hipfftDestroy(plan); 

	free(input);
	free(output);

	hipFree(idata);
	hipFree(odata); 

	return 0;
}

