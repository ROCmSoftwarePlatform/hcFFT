#include <stdio.h>
#include <assert.h>
// CUDA runtime
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#define NX 256 
#define NY 128  
#define BATCH 1

int main()
{
	hipfftHandle plan;
	int Csize = NX * NY;
	hipfftComplex *input = (hipfftComplex*)calloc(Csize, sizeof(hipfftComplex));
	hipfftComplex *output = (hipfftComplex*)calloc(Csize, sizeof(hipfftComplex));
	hipfftComplex *idata;
	hipfftComplex *odata;

	int seed = 123456789;
	srand(seed);

	for(int i=0; i<Csize; i++)
	{
		input[i].x = rand();
		input[i].y = rand();
	}

	hipMalloc((void**)&idata, sizeof(hipfftComplex)*Csize*BATCH);
	hipMemcpy(idata, input, sizeof(hipfftComplex)*Csize*BATCH, hipMemcpyHostToDevice);
	hipMalloc((void**)&odata, sizeof(hipfftComplex)*Csize*BATCH);
	hipMemcpy(odata, output, sizeof(hipfftComplex)*Csize*BATCH, hipMemcpyHostToDevice);
 
	if (hipGetLastError() != hipSuccess)
	{ 
		fprintf(stderr, "Cuda error: Failed to allocate\n"); 
		return 0;
	} 

	/* Create a 2D FFT plan. */ 
	if (hipfftPlan2d(&plan, NX, NY, HIPFFT_C2C) != HIPFFT_SUCCESS)
	{ 
		fprintf(stderr, "CUFFT Error: Unable to create plan\n"); 
		return 0;
	}

	if (hipfftExecC2C(plan, (hipfftComplex*)idata, (hipfftComplex*)odata, HIPFFT_FORWARD) != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT error: ExecC2C Forward failed");
		return 0;	
	}

	if (hipfftExecC2C(plan, (hipfftComplex*)idata, (hipfftComplex*)odata, HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT error: ExecC2C Inverse failed");
		return 0;	
	}

	if (hipDeviceSynchronize() != hipSuccess)
	{ 
		fprintf(stderr, "Cuda error: Failed to synchronize\n"); 
		return 0; 
	}

	hipfftDestroy(plan); 
	
	free(input);
	free(output);

	hipFree(idata);
	hipFree(odata);

	return 0;
}

